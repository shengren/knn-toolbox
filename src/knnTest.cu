#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "../include/cuknns.h"


void rand_init(knntype *a, int N){

  srand(time(NULL));
  for(int i=0; i<N; i++){
    a[i] = 100 * (knntype)rand() / RAND_MAX - 50;
  }

}

void load(knntype *a, char *file, int N){

  FILE* infile;
  size_t res;

  printf("Loading data from file: %s\n", file);

  if((infile=fopen(file, "rb"))==NULL){
    printf("Can't open input file\n");
  }

  res = fread(a, sizeof(knntype), N, infile);

  fclose(infile);

}

void save(knntype *a, char *file, int N){

  FILE* outfile;

  printf("Saving data to file: %s\n", file);

  if((outfile=fopen(file, "wb"))==NULL){
    printf("Can't open output file");
  }

  fwrite(a, sizeof(knntype), N, outfile);

  fclose(outfile);

}

void serial_dot(knntype *dot, knntype *data, int N, int D){

  for(int i=0; i<N; i++){
    knntype tmp= 0;
    for(int j=0; j<D; j++){
      knntype tt = data[i*D + j];
      tmp += tt * tt;
    }
    dot[i] = tmp;
  }

}

int main(int argc, char** argv){
  assert(argc == 8);

  char *datafile = argv[1];
  char *queryfile = argv[2];

  char *distfile = "KNNdist.bin";
  char *idxfile = "KNNidx.bin";


  long int N = atoi(argv[3]);
  long int Q = atoi(argv[4]);
  long int D = atoi(argv[5]);
  long int k = atoi(argv[6]);
  int alg = atoi(argv[7]);

  knntype *data, *queries, *KNNdist, *KNNidx, *dp;
  hipHostAlloc((void**)&data, N*D*sizeof(knntype), hipHostMallocWriteCombined);
  hipHostAlloc((void**)&queries, Q*D*sizeof(knntype), hipHostMallocWriteCombined);
  KNNdist = (knntype*)malloc(Q*k*sizeof(knntype));
  KNNidx = (knntype*)malloc(Q*k*sizeof(knntype));
  hipHostAlloc((void**)&dp, N*sizeof(knntype), hipHostMallocWriteCombined);


  knntype *finalDist = (knntype*)malloc(k*Q*sizeof(knntype));
  knntype *finalIdx = (knntype*)malloc(k*Q*sizeof(knntype));

  printf("files: %s %s\n", datafile, queryfile);

  load(data, datafile, N*D);
  load(queries, queryfile, Q*D);

  knnplan plan;

  //knnsplan(&plan, N, Q, D, k);

  plan.objects = N;
  plan.numQueries = Q;
  plan.dimentions = D;
  plan.k = k;
  plan.numStreams = 1;
  if (alg == 0)
    plan.pt2Function = &gpuknnsBitonic;
  else if (alg == 1)
    plan.pt2Function = &gpuknnsHeap;
  else
    exit(-1);

  knnsexecute(plan, data, queries, KNNdist, KNNidx);

  save(KNNdist, distfile, k*Q);
  save(KNNidx, idxfile, k*Q);

  hipHostFree(data);
  hipHostFree(queries);
  hipHostFree(KNNdist);
  hipHostFree(KNNidx);
  hipHostFree(dp);
  free(finalDist);
  free(finalIdx);

}
