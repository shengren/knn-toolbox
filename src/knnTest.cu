#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "../include/cuknns.h"


void rand_init(knntype *a, int N){

  srand(time(NULL));
  for(int i=0; i<N; i++){
    a[i] = 100 * (knntype)rand() / RAND_MAX - 50;
  }

}

void load(knntype *a, char *file, int N){

  FILE* infile;
  size_t res;

  printf("Loading data from file: %s\n", file);

  if((infile=fopen(file, "rb"))==NULL){
    printf("Can't open input file\n");
  }

  res = fread(a, sizeof(knntype), N, infile);

  fclose(infile);

}

void save(knntype *a, char *file, int N){

  FILE* outfile;

  printf("Saving data to file: %s\n", file);

  if((outfile=fopen(file, "wb"))==NULL){
    printf("Can't open output file");
  }

  fwrite(a, sizeof(knntype), N, outfile);

  fclose(outfile);

}

void serial_dot(knntype *dot, knntype *data, int N, int D){

  for(int i=0; i<N; i++){
    knntype tmp= 0;
    for(int j=0; j<D; j++){
      knntype tt = data[i*D + j];
      tmp += tt * tt;
    }
    dot[i] = tmp;
  }

}

int main(int argc, char** argv){
  assert(argc == 8);

  char *datafile = argv[1];
  char *queryfile = argv[2];

  char *distfile = "KNNdist.bin";
  char *idxfile = "KNNidx.bin";


  long int N = atoi(argv[3]);
  long int Q = atoi(argv[4]);
  long int D = atoi(argv[5]);
  long int k = atoi(argv[6]);
  int alg = atoi(argv[7]);

  knntype *data, *queries, *KNNdist, *KNNidx, *dp;
  hipHostAlloc((void**)&data, N*D*sizeof(knntype), hipHostMallocWriteCombined);
  hipHostAlloc((void**)&queries, Q*D*sizeof(knntype), hipHostMallocWriteCombined);
  KNNdist = (knntype*)malloc(Q*k*sizeof(knntype));
  KNNidx = (knntype*)malloc(Q*k*sizeof(knntype));
  hipHostAlloc((void**)&dp, N*sizeof(knntype), hipHostMallocWriteCombined);


  knntype *finalDist = (knntype*)malloc(k*Q*sizeof(knntype));
  knntype *finalIdx = (knntype*)malloc(k*Q*sizeof(knntype));

  printf("files: %s %s\n", datafile, queryfile);

  knntype *buf;

  buf = (knntype *)malloc(N * D * sizeof(knntype));
  load(buf, datafile, N*D);
  for (int i = 0; i < N; ++i) {
    for (int j = 0; j < D; ++j) {
      data[i * D + j] = buf[j * N + i];
    }
  }
  free(buf);

  buf = (knntype *)malloc(Q * D * sizeof(knntype));
  load(buf, queryfile, Q*D);
  for (int i = 0; i < Q; ++i) {
    for (int j = 0; j < D; ++j) {
      queries[i * D + j] = buf[j * Q + i];
    }
  }
  free(buf);

  knnplan plan;

  //knnsplan(&plan, N, Q, D, k);

  plan.objects = N;
  plan.numQueries = Q;
  plan.dimentions = D;
  plan.k = k;
  plan.numStreams = 1;
  if (alg == 0)
    plan.pt2Function = &gpuknnsBitonic;
  else if (alg == 1)
    plan.pt2Function = &gpuknnsHeap;
  else
    exit(EXIT_FAILURE);

  knnsexecute(plan, data, queries, KNNdist, KNNidx);

  //save(KNNdist, distfile, k*Q);
  //save(KNNidx, idxfile, k*Q);

  FILE *file_knn_dist = fopen("dist.txt", "w");
  for (int i = 0; i < Q; ++i) {
    for (int j = 0; j < k; ++j) {
      if (j > 0) fprintf(file_knn_dist, " ");
      fprintf(file_knn_dist, "%.5f", KNNdist[j * Q + i]);
    }
    fprintf(file_knn_dist, "\n");
  }
  fclose(file_knn_dist);

  FILE *file_knn_idx = fopen("idx.txt", "w");
  for (int i = 0; i < Q; ++i) {
    for (int j = 0; j < k; ++j) {
      if (j > 0) fprintf(file_knn_idx, " ");
      fprintf(file_knn_idx, "%.0f", KNNidx[j * Q + i]);
    }
    fprintf(file_knn_idx, "\n");
  }
  fclose(file_knn_idx);

  hipHostFree(data);
  hipHostFree(queries);
  hipHostFree(KNNdist);
  hipHostFree(KNNidx);
  hipHostFree(dp);
  free(finalDist);
  free(finalIdx);

}
